#include "hip/hip_runtime.h"
#include "btcusdt.cuh"

#include "../impl_template/tmpl_etc.cu"

BTCUSDT_t * cree_btcusdt() {
	//
	BTCUSDT_t * ret = (BTCUSDT_t*)malloc(sizeof(BTCUSDT_t));

	//
	FILE * fp = fopen("prixs/dar.bin", "rb");
	ASSERT(fp != 0);
	FREAD(&ret->T, sizeof(uint), 1, fp);
	
	//
	uint LIGNES, N, P;
	FREAD(&LIGNES, sizeof(uint), 1, fp);
	FREAD(&N,      sizeof(uint), 1, fp);
	FREAD(&P,      sizeof(uint), 1, fp);

	//
	ret->X = N * LIGNES;
	ret->Y = P;

	//
	float * x = alloc<float>(ret->T * ret->X);
	FREAD(x, sizeof(float), ret->T * ret->X, fp);
	ret->entree = espace_cree(ret->X, ret->T);
	CONTROLE_CUDA(hipMemcpy(
		ret->entree->y__d,
		x,
		sizeof(float)*ret->T*ret->X,
		hipMemcpyHostToDevice
	));
	free(x);

	//
	float * y = alloc<float>(ret->T * ret->Y);
	FREAD(y, sizeof(float), ret->T * ret->Y, fp);
	ret->sortie = espace_cree(ret->Y, ret->T);
	CONTROLE_CUDA(hipMemcpy(
		ret->sortie->y__d,
		y,
		sizeof(float)*ret->T*ret->Y,
		hipMemcpyHostToDevice
	));
	free(y);

	//
	fclose(fp);

	//
	return ret;
};

void liberer_btcusdt(BTCUSDT_t * donnee) {
	espace_liberer(donnee->entree);
	espace_liberer(donnee->sortie);
};

//	====================================================

static __global__ void k__pourcent_btcusdt(
	float * somme, float * potentiel,
	float * y, float * p1p0,
	float coef_puissance,
	uint * ts__d,
	uint P)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	uint p      = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (t < GRAND_T && mega_t < MEGA_T && p < P) {
		uint _t = ts__d[t] + mega_t;
		//
		uint a_t_il_predit = (sng(p1p0[_t*P + p]) == sng(y[_t*P + p]));
		//
		float _____somme = powf(fabs(p1p0[_t*P + p]), coef_puissance) * a_t_il_predit;
		float _potentiel = powf(fabs(p1p0[_t*P + p]), coef_puissance) * true         ;
		//
		atomicAdd(&somme    [p], _____somme);
		atomicAdd(&potentiel[p], _potentiel);
	}
};

float *  pourcent_btcusdt(BTCUSDT_t * btcusdt, Espace_t * y, uint * ts__d, float coef_puissance) {
	uint P = btcusdt->Y;
	//
	float *     somme__d = cudalloc<float>(P);
	float * potentiel__d = cudalloc<float>(P);
	//
	k__pourcent_btcusdt<<<dim3(KERD(GRAND_T, 32), KERD(MEGA_T, 32), KERD(P, 4)), dim3(32,32,4)>>>(
		somme__d, potentiel__d,
		y->y__d, btcusdt->sortie->y__d,
		coef_puissance,
		ts__d,
		P
	);
	ATTENDRE_CUDA();
	//
	float * somme     = gpu_vers_cpu<float>(    somme__d, P);
	float * potentiel = gpu_vers_cpu<float>(potentiel__d, P);
	//
	float * ret = alloc<float>(P);
	FOR(0, p, P) ret[p] = somme[p] / potentiel[p];
	//
	cudafree<float>(    somme__d);
	cudafree<float>(potentiel__d);
	    free(           somme   );
	    free(       potentiel   );
	//
	return ret;
};

//	====================================================

static __global__ void k__f_btcusdt(
	float * somme_score,
	float * y, float * p1p0,
	uint * ts__d,
	uint P)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	uint p      = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (t < GRAND_T && mega_t < MEGA_T && p < P) {
		uint _t = ts__d[t] + mega_t;
		//
		atomicAdd(&somme_score[0], S(y[_t*P + p], p1p0[_t*P + p]));
	}
};

float f_btcusdt(BTCUSDT_t * btcusdt, Espace_t * y, uint * ts__d) {
	uint P = btcusdt->Y;
	//
	float * somme__d = cudalloc<float>(1);
	//
	k__f_btcusdt<<<dim3(KERD(GRAND_T, 32), KERD(MEGA_T, 32), KERD(P, 4)), dim3(32,32,4)>>>(
		somme__d,
		y->y__d, btcusdt->sortie->y__d,
		ts__d,
		P
	);
	ATTENDRE_CUDA();
	//
	float * somme = gpu_vers_cpu<float>(somme__d, 1);
	//
	float score = somme[0] / ((float)(P * GRAND_T * MEGA_T));
	//
	cudafree<float>(somme__d);
	    free       (somme   );
	//
	return score;
};

//	====================================================

static __global__ void k__df_btcusdt(
	float * y, float * p1p0, float * dy,
	uint * ts__d,
	uint P)
{
	uint t      = threadIdx.x + blockIdx.x * blockDim.x;
	uint mega_t = threadIdx.y + blockIdx.y * blockDim.y;
	uint p      = threadIdx.z + blockIdx.z * blockDim.z;
	//
	if (t < GRAND_T && mega_t < MEGA_T && p < P) {
		uint _t = ts__d[t] + mega_t;
		//
		atomicAdd(&dy[_t*P + p], dS(y[_t*P + p], p1p0[_t*P + p]) / (float)(P * MEGA_T * GRAND_T));
	}
};

void df_btcusdt(BTCUSDT_t * btcusdt, Espace_t * y, uint * ts__d) {
	uint P = btcusdt->Y;
	//
	k__df_btcusdt<<<dim3(KERD(GRAND_T, 32), KERD(MEGA_T, 32), KERD(P, 4)), dim3(32,32,4)>>>(
		y->y__d, btcusdt->sortie->y__d, y->dy__d,
		ts__d,
		P
	);
	ATTENDRE_CUDA();
};